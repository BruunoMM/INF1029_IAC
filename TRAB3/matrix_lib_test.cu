#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "matrix_lib.h"

struct matrix *readDatFile(FILE *arq, int height, int width);
void writeDatFile(FILE *arq, struct matrix *matrix);
struct matrix* createMatrixC(int height, int width);
void freeMatrix(struct matrix* matrix);
void safeCudaMemCpy(float *d_x, float *h_x, int size);
void safeCudaMalloc(float **ptr, int size);

int main(int argc, char *argv[]) {
    FILE *arq, *arq2, *arq3, *arq4;
    int height1, width1, height2, width2;
    float cons;
    char *pathFile1, *pathFile2, *pathResultFile1, *pathResultFile2;

    cons = atof(argv[1]);
    height1 = atoi(argv[2]);
    width1 = atoi(argv[3]);
    height2 = atoi(argv[4]);
    width2 = atoi(argv[5]);
    pathFile1 = argv[6];
    pathFile2 = argv[7];
    pathResultFile1 = argv[8];
    pathResultFile2 = argv[9];

    if(argc != 10) {
        printf("Numero de argumentos invalidos.\n");
        return -1;
    } else if ((height1 + width1 + height2 + width2) % 8 != 0) {
        printf("Dimensoes nao multiplas de 8.\n");
        return -1;
    }
    
    arq = fopen(pathFile1, "rb");
    arq2 = fopen(pathFile2, "rb");
    arq3 = fopen(pathResultFile1, "wb");
    arq4 = fopen(pathResultFile2, "wb");
    
    if (!arq || !arq2) {
        printf("Arquivos de entrada nao existentes.\n");
        return -1;
    }

    if(!arq3 || !arq4) {
        printf("Nao foi possivel criar arquivos de saida.\n");
        return -1;
    }
    struct matrix *matrixA = readDatFile(arq, height1, width1);
    // struct matrix *matrixB = readDatFile(arq2, height2, width2);
    // struct matrix *matrixC = createMatrixC(height1, width2);

    int blockSize = THREADS_PER_BLOCK;
    int numBlocks = (height1 * width1 + blockSize - 1) / blockSize;
    
    mult_scalar<<<numBlocks, blockSize>>>(cons, height1 * width1, matrixA);
    printMatrix(matrixA);

    // freeMatrix(matrixA);
    // freeMatrix(matrixB);
    // freeMatrix(matrixC);

    fclose(arq);    
    fclose(arq2);   
    fclose(arq3);
    fclose(arq4);
    
    return 0;
}

// struct matrix* createMatrixC(int height, int width) {
//     struct matrix *matrixC = malloc(sizeof(struct matrix));
//     matrixC->h_rows = aligned_alloc(32, height * width * sizeof(float));
//     matrixC->height = height;
//     matrixC->width = width;

//     return matrixC;
// }

void freeMatrix(struct matrix* matrix) {
    free(matrix->h_rows);
    free(matrix);
}

struct matrix *readDatFile(FILE *arq, int height, int width){
    struct matrix *matrixEx;
    float *rows;
    int count = 0;
    int totalSize = height * width;
    
    matrixEx = (struct matrix *) malloc(sizeof(struct matrix));
    safeCudaMalloc(&matrixEx->d_rows, totalSize);
    rows = (float *)malloc(totalSize*sizeof(float));

    matrixEx->height = height;
    matrixEx->width = width;

    while (!feof(arq) && (count < totalSize)){
        fread(&rows[count], sizeof(float), 1, arq);
        count++;
    }

    matrixEx->h_rows = rows;
    safeCudaMemCpy(matrixEx->d_rows, matrixEx->h_rows, totalSize);

    return matrixEx;
}

void safeCudaMemCpy(float *d_x, float *h_x, int size) {
    hipError_t cudaResult;
    cudaResult = hipMemcpy(d_x, h_x, size*sizeof(float), hipMemcpyHostToDevice);

    if (cudaResult != hipSuccess) {
	    printf("hipMemcpy (h_x -> d_x) returned error %s (code %d), line(%d)\n", hipGetErrorString(cudaResult), cudaResult, __LINE__);
        exit(1);
    }
}

void safeCudaMalloc(float **ptr, int size) {
    hipError_t cudaResult; 
    cudaResult = hipMalloc(ptr, size* sizeof(float));
    if (cudaResult != hipSuccess) {
	    printf("hipMalloc d_x returned error %s (code %d)\n", hipGetErrorString(cudaResult), cudaResult);
        exit(1);
    }
}

void writeDatFile(FILE *arq, struct matrix *matrix) {
    int height = matrix->height;
    int width = matrix->width;
    int totalSize = height * width;

    for(int i=0 ; i < totalSize ; i++) {
        float numberToWrite = matrix->h_rows[i];
        int result = fwrite(&numberToWrite, sizeof(float), 1, arq);
        if(result != 1) {
            printf("Erro de escrita!!! \n");
        }
    }
}
